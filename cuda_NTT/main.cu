#include <iostream>
using std::cout;  
using std::endl;  
#include <chrono>
#include "ntt.cuh"
#include "util.h"
typedef std::chrono::high_resolution_clock Clock;

#define check 1
int main()
{
    unsigned n = 65536*(1<<9);
    int size_array = sizeof(unsigned) * n;
    int size = sizeof(unsigned);
    unsigned q = 469762049, psi = 4782969, psiinv = 392193156;
    // unsigned q = 8380417, psi=1921994, psiinv=527981
    // s = 13, t = 1023 
    unsigned int q_bit = 29;
    // unsinged int q_bit = 23
    unsigned* psiTable = (unsigned*)malloc(size_array);
    unsigned* psiinvTable = (unsigned*)malloc(size_array);
    unsigned* psi_powers, * psiinv_powers;
    fillTablePsi64(psi, q, psiinv, psiTable, psiinvTable, n);
    
    hipMalloc(&psi_powers, size_array);
    hipMalloc(&psiinv_powers, size_array);
    
    hipMemcpy(psi_powers, psiTable, size_array, hipMemcpyHostToDevice);
    hipMemcpy(psiinv_powers, psiinvTable, size_array, hipMemcpyHostToDevice);
    auto t1 = Clock::now();
    
    cout << "q = " << q << endl;
    cout << "root of unity = " << psi << endl;

   
    unsigned int bit_length = q_bit;
    double mu1 = powl(2, 2 * bit_length);
    unsigned mu = mu1 / q;

    unsigned* a;
    hipHostMalloc(&a, sizeof(unsigned) * n);
    randomArray64(a, n, q);
    unsigned* res_a;
    hipHostMalloc(&res_a, sizeof(unsigned) * n);

    unsigned* d_a;
    hipMalloc(&d_a, size_array);

    hipMemcpyAsync(d_a, a, size_array, hipMemcpyHostToDevice, 0);

    // block num 1
    // thread num 1024
    // shared memery 2048*sizeof(unsigned)
    CTBasedNTTInnerSingle<8*(1<<9), 65536*(1<<9)><<<8*(1<<9), 65536*(1<<9), 8192 * sizeof(unsigned), 0>>>(d_a, q, mu, bit_length, psi_powers);
    unsigned* mid_a;
    hipHostMalloc(&mid_a, sizeof(unsigned) * n);
    hipMemcpyAsync(mid_a, d_a, size_array, hipMemcpyDeviceToHost, 0);
    hipDeviceSynchronize(); 
    GSBasedINTTInnerSingle<8*(1<<9), 65536*(1<<9)><<<8*(1<<9), 65536*(1<<9), 8192 * sizeof(unsigned), 0>>>(d_a, q, mu, bit_length, psiinv_powers);

    hipMemcpyAsync(res_a, d_a, size_array, hipMemcpyDeviceToHost, 0);  // do this in async
    hipDeviceSynchronize();  // CPU being a gentleman, and waiting for GPU to finish it's job
    auto t2 = Clock::now();
    printf("Device FFT took %ld \n",
           std::chrono::duration_cast<
                   std::chrono::milliseconds>(t2 - t1)
                   .count());
    bool correct = 1;
    if (check) //check the correctness of results
    {
        for (int i = 0; i < n; i++)
        {
            if (a[i] != res_a[i])
            {
                correct = 0;
                break;
            }
        }
    }

    if (correct)
        cout << "\nNTT and INTT are working correctly." << endl;
    else
        cout << "\nNTT and INTT are not working correctly." << endl;

    hipHostFree(a); hipHostFree(res_a);
    hipFree(d_a);
    return 0;
}


