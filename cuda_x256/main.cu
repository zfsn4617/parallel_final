#include "hip/hip_runtime.h"
#include <stdio.h>
#include "functions.h"
__global__ void hello_from_gpu()
{
    printf("hello word from the gpu %d, %d!\n", blockIdx.x, threadIdx.x);
}

int main()
{
    Initialize_NTT();
    Initialize_iNTT();

    int64 *a = new int64[50000 * 256];
    int64 *b = new int64[50000 * 256];
    int64 *z = new int64[50000 * 256];

    int64 *gpu_a;
    int64 *gpu_b;
    int64 *gpu_a_ntt;
    int64 *gpu_b_ntt;
    int64 *gpu_z_ntt;
    int64 *gpu_z;

    hipMalloc(&gpu_a, 50000 * 256 * sizeof(int64));
    hipMalloc(&gpu_b, 50000 * 256 * sizeof(int64));
    hipMalloc(&gpu_a_ntt, 50000 * 256 * sizeof(int64));
    hipMalloc(&gpu_b_ntt, 50000 * 256 * sizeof(int64));
    hipMalloc(&gpu_z_ntt, 50000 * 256 * sizeof(int64));
    hipMalloc(&gpu_z, 50000 * 256 * sizeof(int64));

    for (int i = 0; i < 50000; i++)
    {
        for (int j = 0; j < 256; j++)
        {
            a[i * 256 + j] = i+1;
            b[i * 256 + j] = i+1;
        }
    }

    auto ckpt0 = high_resolution_clock::now();
    hipMemcpy(gpu_a, a, 50000 * 256 * sizeof(int64), hipMemcpyHostToDevice);
    hipMemcpy(gpu_b, b, 50000 * 256 * sizeof(int64), hipMemcpyHostToDevice);
    auto ckpt1 = high_resolution_clock::now();

    NTT<<<50000, 32>>>(gpu_a, gpu_a_ntt);
    NTT<<<50000, 32>>>(gpu_b, gpu_b_ntt);
    hipDeviceSynchronize();
    hipError_t err;
    err = hipGetLastError();
    printf("%s\n", hipGetErrorString(err));
    auto ckpt2 = high_resolution_clock::now();

    scalar_mul<<<50000, 256>>>(gpu_z_ntt, gpu_a_ntt, gpu_b_ntt);
    hipDeviceSynchronize();
    auto ckpt3 = high_resolution_clock::now();

    iNTT<<<50000, 32>>>(gpu_z_ntt, gpu_z);
    hipDeviceSynchronize();
    auto ckpt4 = high_resolution_clock::now();

    hipMemcpy(z, gpu_z, 50000 * 256 * sizeof(int64), hipMemcpyDeviceToHost);
    auto ckpt5 = high_resolution_clock::now();
    for (int i = 256 * 100; i < 256 * 101; i++)
    {
        cout << setw(8) << z[i];
        if (i % 8 == 7)
            cout << endl;
    }
    double t0 = duration_cast<microseconds>(ckpt1 - ckpt0).count() / 50000.0;
    double t1 = duration_cast<microseconds>(ckpt2 - ckpt1).count() / 50000.0;
    double t2 = duration_cast<microseconds>(ckpt3 - ckpt2).count() / 50000.0;
    double t3 = duration_cast<microseconds>(ckpt4 - ckpt3).count() / 50000.0;
    double t4 = duration_cast<microseconds>(ckpt5 - ckpt4).count() / 50000.0;
    cout << "计算时间0：" << t0 << "微秒/次" << endl;
    cout << "计算时间1：" << t1 << "微秒/次" << endl;
    cout << "计算时间2：" << t2 << "微秒/次" << endl;
    cout << "计算时间3：" << t3 << "微秒/次" << endl;
    cout << "计算时间4：" << t4 << "微秒/次" << endl;
    cout << "NTT乘法计算时间：" << t1 + t2 + t3 << "微秒/次" << endl;
    cout << "总计算时间：" << t0 + t1 + t2 + t3 + t4 << "微秒/次" << endl;
    hipDeviceSynchronize();

    return 0;
}
