#include "hip/hip_runtime.h"
#include "functions.h"
__global__ void scalar_mul(int64 *ret, int64 *a, int64 *b)
{
    int64 *vec_1 = a + blockIdx.x * 256;
    int64 *vec_2 = b + blockIdx.x * 256;
    int64 *vec_ret = ret + blockIdx.x * 256;
    int idx = threadIdx.x;
    vec_ret[idx] = vec_1[idx] * vec_2[idx] % 8380417;
}