#include "hip/hip_runtime.h"
#include "functions.h"

__constant__ int bit_reverse[256];
__constant__ int omega_pow_use_rank[512];

void Initialize_NTT()
{
    hipError_t err;
    int bit_reverse_cpu[256] = {
        0, 128, 64, 192, 32, 160, 96, 224, 16, 144, 80, 208, 48, 176, 112, 240,
        8, 136, 72, 200, 40, 168, 104, 232, 24, 152, 88, 216, 56, 184, 120, 248,
        4, 132, 68, 196, 36, 164, 100, 228, 20, 148, 84, 212, 52, 180, 116, 244,
        12, 140, 76, 204, 44, 172, 108, 236, 28, 156, 92, 220, 60, 188, 124, 252,
        2, 130, 66, 194, 34, 162, 98, 226, 18, 146, 82, 210, 50, 178, 114, 242,
        10, 138, 74, 202, 42, 170, 106, 234, 26, 154, 90, 218, 58, 186, 122, 250,
        6, 134, 70, 198, 38, 166, 102, 230, 22, 150, 86, 214, 54, 182, 118, 246,
        14, 142, 78, 206, 46, 174, 110, 238, 30, 158, 94, 222, 62, 190, 126, 254,
        1, 129, 65, 193, 33, 161, 97, 225, 17, 145, 81, 209, 49, 177, 113, 241,
        9, 137, 73, 201, 41, 169, 105, 233, 25, 153, 89, 217, 57, 185, 121, 249,
        5, 133, 69, 197, 37, 165, 101, 229, 21, 149, 85, 213, 53, 181, 117, 245,
        13, 141, 77, 205, 45, 173, 109, 237, 29, 157, 93, 221, 61, 189, 125, 253,
        3, 131, 67, 195, 35, 163, 99, 227, 19, 147, 83, 211, 51, 179, 115, 243,
        11, 139, 75, 203, 43, 171, 107, 235, 27, 155, 91, 219, 59, 187, 123, 251,
        7, 135, 71, 199, 39, 167, 103, 231, 23, 151, 87, 215, 55, 183, 119, 247,
        15, 143, 79, 207, 47, 175, 111, 239, 31, 159, 95, 223, 63, 191, 127, 255};
    hipMemcpyToSymbol(HIP_SYMBOL(bit_reverse), bit_reverse_cpu, sizeof(bit_reverse_cpu));
    err = hipGetLastError();
    printf("%s\n", hipGetErrorString(err));
    int omega_pow_use_rank_cpu[512] = {
        0, 8380416, 4808194, 3572223, 4614810, 4618904, 3765607, 3761513,
        2883726, 5178987, 5178923, 3145678, 5496691, 3201430, 3201494, 5234739,
        6250525, 7822959, 601683, 7375178, 2682288, 1221177, 4837932, 4615550,
        2129892, 557458, 7778734, 1005239, 5698129, 7159240, 3542485, 3764867,
        7044481, 4795319, 4317364, 2453983, 4855975, 6096684, 1674615, 6666122,
        7703827, 642628, 3370349, 1460718, 7946292, 2815639, 2663378, 5152541,
        1335936, 3585098, 4063053, 5926434, 3524442, 2283733, 6705802, 1714295,
        676590, 7737789, 5010068, 6919699, 434125, 5564778, 5717039, 3227876,
        3241972, 7823561, 2740543, 4623627, 394148, 1858416, 7220542, 4805951,
        4018989, 3192354, 5197539, 6663429, 7284949, 2917338, 3110818, 3415069,
        2156050, 4510100, 4793971, 1935799, 928749, 5034454, 3704823, 817536,
        2071829, 2897314, 3602218, 4430364, 3506380, 1853806, 6279007, 1759347,
        5138445, 556856, 5639874, 3756790, 7986269, 6522001, 1159875, 3574466,
        4361428, 5188063, 3182878, 1716988, 1095468, 5463079, 5269599, 4965348,
        6224367, 3870317, 3586446, 6444618, 7451668, 3345963, 4675594, 7562881,
        6308588, 5483103, 4778199, 3950053, 4874037, 6526611, 2101410, 6621070,
        6644104, 6067579, 4183372, 2461387, 6852351, 2236726, 4222329, 7080401,
        5183169, 5697147, 4528402, 3901472, 169688, 8031605, 6352299, 5801164,
        5130263, 7921254, 3121440, 7759253, 1148858, 6458164, 5569126, 4182915,
        4213992, 5604662, 5307408, 5454601, 3334383, 1011223, 4564692, 2391089,
        8145010, 4912752, 5157610, 1317678, 7897768, 6635910, 7270901, 6018354,
        6392603, 2778788, 5744944, 7153756, 565603, 327848, 2508980, 1787943,
        3258457, 653275, 274060, 3035980, 5418153, 3818627, 2983781, 3482206,
        4892034, 7023969, 7102792, 5006167, 2462444, 6026202, 6442847, 2254727,
        1736313, 2312838, 4197045, 5919030, 1528066, 6143691, 4158088, 1300016,
        3197248, 2683270, 3852015, 4478945, 8210729, 348812, 2028118, 2579253,
        3250154, 459163, 5258977, 621164, 7231559, 1922253, 2811291, 4197502,
        4166425, 2775755, 3073009, 2925816, 5046034, 7369194, 3815725, 5989328,
        235407, 3467665, 3222807, 7062739, 482649, 1744507, 1109516, 2362063,
        1987814, 5601629, 2635473, 1226661, 7814814, 8052569, 5871437, 6592474,
        5121960, 7727142, 8106357, 5344437, 2962264, 4561790, 5396636, 4898211,
        3488383, 1356448, 1277625, 3374250, 5917973, 2354215, 1937570, 6125690,
        1921994, 7826699, 1182243, 5732423, 6607829, 781875, 5925040, 507927,
        1310261, 214880, 5607817, 4399818, 1239911, 5256655, 5926272, 6757063,
        6341273, 140244, 2296397, 4357667, 2387513, 3974485, 4969849, 1393159,
        5382198, 7009900, 1935420, 2028038, 12417, 3014420, 4423473, 1179613,
        4908348, 3105558, 7743490, 8041997, 1727088, 7648983, 4829411, 724804,
        613238, 770441, 5720009, 6764887, 6084318, 6187330, 8352605, 2374402,
        7561656, 4949981, 4663471, 5767564, 268456, 3531229, 3768948, 1476985,
        8291116, 11879, 6924527, 3369273, 5184741, 2926054, 6783595, 5637006,
        7921677, 7872272, 87208, 5370669, 4146264, 1900052, 250446, 7192532,
        2218467, 5016875, 8321269, 5811406, 4541938, 6195333, 7371052, 2105286,
        1879878, 6866265, 4423672, 7630840, 4768667, 3773731, 1685153, 2491325,
        8238582, 3020393, 1753, 6715099, 1254190, 1716814, 4620952, 586241,
        4340221, 7277073, 3965306, 3033742, 2192938, 7325939, 635956, 1834526,
        1354892, 545376, 1780227, 1723229, 3747250, 6022044, 822541, 2033807,
        6201452, 860144, 3284915, 4148469, 3180456, 303005, 2678278, 6386371,
        2513018, 3994671, 2659525, 1163598, 5737437, 7987710, 6400920, 7852436,
        6458423, 553718, 7198174, 2647994, 1772588, 7598542, 2455377, 7872490,
        7070156, 8165537, 2772600, 3980599, 7140506, 3123762, 2454145, 1623354,
        2039144, 8240173, 6084020, 4022750, 5992904, 4405932, 3410568, 6987258,
        2998219, 1370517, 6444997, 6352379, 8368000, 5365997, 3956944, 7200804,
        3472069, 5274859, 636927, 338420, 6653329, 731434, 3551006, 7655613,
        7767179, 7609976, 2660408, 1615530, 2296099, 2193087, 27812, 6006015,
        818761, 3430436, 3716946, 2612853, 8111961, 4849188, 4611469, 6903432,
        89301, 8368538, 1455890, 5011144, 3195676, 5454363, 1596822, 2743411,
        458740, 508145, 8293209, 3009748, 4234153, 6480365, 8129971, 1187885,
        6161950, 3363542, 59148, 2569011, 3838479, 2185084, 1009365, 6275131,
        6500539, 1514152, 3956745, 749577, 3611750, 4606686, 6695264, 5889092,
        141835, 5360024, 8378664, 1665318, 7126227, 6663603, 3759465, 7794176,
        4040196, 1103344, 4415111, 5346675, 6187479, 1054478, 7744461, 6545891,
        7025525, 7835041, 6600190, 6657188, 4633167, 2358373, 7557876, 6346610,
        2178965, 7520273, 5095502, 4231948, 5199961, 8077412, 5702139, 1994046,
        5867399, 4385746, 5720892, 7216819, 2642980, 392707, 1979497, 527981};
    hipMemcpyToSymbol(HIP_SYMBOL(omega_pow_use_rank), omega_pow_use_rank_cpu, sizeof(omega_pow_use_rank_cpu));
    err = hipGetLastError();
    printf("%s\n", hipGetErrorString(err));
}

__global__ void NTT(int64 *a, int64 *out)
{
    int64 *vec_1 = a + blockIdx.x * 256;
    int64 *vec_2 = out + blockIdx.x * 256;
    int idx = threadIdx.x;
    int idx0 = idx;
    int idx1 = idx + 32;
    int idx2 = idx + 64;
    int idx3 = idx + 96;
    int idx4 = idx + 128;
    int idx5 = idx + 160;
    int idx6 = idx + 192;
    int idx7 = idx + 224;

    vec_2[idx0] = vec_1[bit_reverse[idx0]];
    vec_2[idx1] = vec_1[bit_reverse[idx1]];
    vec_2[idx2] = vec_1[bit_reverse[idx2]];
    vec_2[idx3] = vec_1[bit_reverse[idx3]];
    vec_2[idx4] = vec_1[bit_reverse[idx4]];
    vec_2[idx5] = vec_1[bit_reverse[idx5]];
    vec_2[idx6] = vec_1[bit_reverse[idx6]];
    vec_2[idx7] = vec_1[bit_reverse[idx7]];

    vec_1[idx0] = (vec_2[idx0 & 0xFFFFFFFE] + (int64)omega_pow_use_rank[(idx0 & 0x01) | 0x02] * vec_2[idx0 | 0x00000001]) % 8380417;
    vec_1[idx1] = (vec_2[idx1 & 0xFFFFFFFE] + (int64)omega_pow_use_rank[(idx1 & 0x01) | 0x02] * vec_2[idx1 | 0x00000001]) % 8380417;
    vec_1[idx2] = (vec_2[idx2 & 0xFFFFFFFE] + (int64)omega_pow_use_rank[(idx2 & 0x01) | 0x02] * vec_2[idx2 | 0x00000001]) % 8380417;
    vec_1[idx3] = (vec_2[idx3 & 0xFFFFFFFE] + (int64)omega_pow_use_rank[(idx3 & 0x01) | 0x02] * vec_2[idx3 | 0x00000001]) % 8380417;
    vec_1[idx4] = (vec_2[idx4 & 0xFFFFFFFE] + (int64)omega_pow_use_rank[(idx4 & 0x01) | 0x02] * vec_2[idx4 | 0x00000001]) % 8380417;
    vec_1[idx5] = (vec_2[idx5 & 0xFFFFFFFE] + (int64)omega_pow_use_rank[(idx5 & 0x01) | 0x02] * vec_2[idx5 | 0x00000001]) % 8380417;
    vec_1[idx6] = (vec_2[idx6 & 0xFFFFFFFE] + (int64)omega_pow_use_rank[(idx6 & 0x01) | 0x02] * vec_2[idx6 | 0x00000001]) % 8380417;
    vec_1[idx7] = (vec_2[idx7 & 0xFFFFFFFE] + (int64)omega_pow_use_rank[(idx7 & 0x01) | 0x02] * vec_2[idx7 | 0x00000001]) % 8380417;

    vec_2[idx0] = (vec_1[idx0 & 0xFFFFFFFD] + (int64)omega_pow_use_rank[(idx0 & 0x03) | 0x04] * vec_1[idx0 | 0x00000002]) % 8380417;
    vec_2[idx1] = (vec_1[idx1 & 0xFFFFFFFD] + (int64)omega_pow_use_rank[(idx1 & 0x03) | 0x04] * vec_1[idx1 | 0x00000002]) % 8380417;
    vec_2[idx2] = (vec_1[idx2 & 0xFFFFFFFD] + (int64)omega_pow_use_rank[(idx2 & 0x03) | 0x04] * vec_1[idx2 | 0x00000002]) % 8380417;
    vec_2[idx3] = (vec_1[idx3 & 0xFFFFFFFD] + (int64)omega_pow_use_rank[(idx3 & 0x03) | 0x04] * vec_1[idx3 | 0x00000002]) % 8380417;
    vec_2[idx4] = (vec_1[idx4 & 0xFFFFFFFD] + (int64)omega_pow_use_rank[(idx4 & 0x03) | 0x04] * vec_1[idx4 | 0x00000002]) % 8380417;
    vec_2[idx5] = (vec_1[idx5 & 0xFFFFFFFD] + (int64)omega_pow_use_rank[(idx5 & 0x03) | 0x04] * vec_1[idx5 | 0x00000002]) % 8380417;
    vec_2[idx6] = (vec_1[idx6 & 0xFFFFFFFD] + (int64)omega_pow_use_rank[(idx6 & 0x03) | 0x04] * vec_1[idx6 | 0x00000002]) % 8380417;
    vec_2[idx7] = (vec_1[idx7 & 0xFFFFFFFD] + (int64)omega_pow_use_rank[(idx7 & 0x03) | 0x04] * vec_1[idx7 | 0x00000002]) % 8380417;

    vec_1[idx0] = (vec_2[idx0 & 0xFFFFFFFB] + (int64)omega_pow_use_rank[(idx0 & 0x07) | 0x08] * vec_2[idx0 | 0x00000004]) % 8380417;
    vec_1[idx1] = (vec_2[idx1 & 0xFFFFFFFB] + (int64)omega_pow_use_rank[(idx1 & 0x07) | 0x08] * vec_2[idx1 | 0x00000004]) % 8380417;
    vec_1[idx2] = (vec_2[idx2 & 0xFFFFFFFB] + (int64)omega_pow_use_rank[(idx2 & 0x07) | 0x08] * vec_2[idx2 | 0x00000004]) % 8380417;
    vec_1[idx3] = (vec_2[idx3 & 0xFFFFFFFB] + (int64)omega_pow_use_rank[(idx3 & 0x07) | 0x08] * vec_2[idx3 | 0x00000004]) % 8380417;
    vec_1[idx4] = (vec_2[idx4 & 0xFFFFFFFB] + (int64)omega_pow_use_rank[(idx4 & 0x07) | 0x08] * vec_2[idx4 | 0x00000004]) % 8380417;
    vec_1[idx5] = (vec_2[idx5 & 0xFFFFFFFB] + (int64)omega_pow_use_rank[(idx5 & 0x07) | 0x08] * vec_2[idx5 | 0x00000004]) % 8380417;
    vec_1[idx6] = (vec_2[idx6 & 0xFFFFFFFB] + (int64)omega_pow_use_rank[(idx6 & 0x07) | 0x08] * vec_2[idx6 | 0x00000004]) % 8380417;
    vec_1[idx7] = (vec_2[idx7 & 0xFFFFFFFB] + (int64)omega_pow_use_rank[(idx7 & 0x07) | 0x08] * vec_2[idx7 | 0x00000004]) % 8380417;

    vec_2[idx0] = (vec_1[idx0 & 0xFFFFFFF7] + (int64)omega_pow_use_rank[(idx0 & 0x0F) | 0x10] * vec_1[idx0 | 0x00000008]) % 8380417;
    vec_2[idx1] = (vec_1[idx1 & 0xFFFFFFF7] + (int64)omega_pow_use_rank[(idx1 & 0x0F) | 0x10] * vec_1[idx1 | 0x00000008]) % 8380417;
    vec_2[idx2] = (vec_1[idx2 & 0xFFFFFFF7] + (int64)omega_pow_use_rank[(idx2 & 0x0F) | 0x10] * vec_1[idx2 | 0x00000008]) % 8380417;
    vec_2[idx3] = (vec_1[idx3 & 0xFFFFFFF7] + (int64)omega_pow_use_rank[(idx3 & 0x0F) | 0x10] * vec_1[idx3 | 0x00000008]) % 8380417;
    vec_2[idx4] = (vec_1[idx4 & 0xFFFFFFF7] + (int64)omega_pow_use_rank[(idx4 & 0x0F) | 0x10] * vec_1[idx4 | 0x00000008]) % 8380417;
    vec_2[idx5] = (vec_1[idx5 & 0xFFFFFFF7] + (int64)omega_pow_use_rank[(idx5 & 0x0F) | 0x10] * vec_1[idx5 | 0x00000008]) % 8380417;
    vec_2[idx6] = (vec_1[idx6 & 0xFFFFFFF7] + (int64)omega_pow_use_rank[(idx6 & 0x0F) | 0x10] * vec_1[idx6 | 0x00000008]) % 8380417;
    vec_2[idx7] = (vec_1[idx7 & 0xFFFFFFF7] + (int64)omega_pow_use_rank[(idx7 & 0x0F) | 0x10] * vec_1[idx7 | 0x00000008]) % 8380417;

    vec_1[idx0] = (vec_2[idx0 & 0xFFFFFFEF] + (int64)omega_pow_use_rank[(idx0 & 0x1F) | 0x20] * vec_2[idx0 | 0x00000010]) % 8380417;
    vec_1[idx1] = (vec_2[idx1 & 0xFFFFFFEF] + (int64)omega_pow_use_rank[(idx1 & 0x1F) | 0x20] * vec_2[idx1 | 0x00000010]) % 8380417;
    vec_1[idx2] = (vec_2[idx2 & 0xFFFFFFEF] + (int64)omega_pow_use_rank[(idx2 & 0x1F) | 0x20] * vec_2[idx2 | 0x00000010]) % 8380417;
    vec_1[idx3] = (vec_2[idx3 & 0xFFFFFFEF] + (int64)omega_pow_use_rank[(idx3 & 0x1F) | 0x20] * vec_2[idx3 | 0x00000010]) % 8380417;
    vec_1[idx4] = (vec_2[idx4 & 0xFFFFFFEF] + (int64)omega_pow_use_rank[(idx4 & 0x1F) | 0x20] * vec_2[idx4 | 0x00000010]) % 8380417;
    vec_1[idx5] = (vec_2[idx5 & 0xFFFFFFEF] + (int64)omega_pow_use_rank[(idx5 & 0x1F) | 0x20] * vec_2[idx5 | 0x00000010]) % 8380417;
    vec_1[idx6] = (vec_2[idx6 & 0xFFFFFFEF] + (int64)omega_pow_use_rank[(idx6 & 0x1F) | 0x20] * vec_2[idx6 | 0x00000010]) % 8380417;
    vec_1[idx7] = (vec_2[idx7 & 0xFFFFFFEF] + (int64)omega_pow_use_rank[(idx7 & 0x1F) | 0x20] * vec_2[idx7 | 0x00000010]) % 8380417;

    vec_2[idx0] = (vec_1[idx0 & 0xFFFFFFDF] + (int64)omega_pow_use_rank[(idx0 & 0x3F) | 0x40] * vec_1[idx0 | 0x00000020]) % 8380417;
    vec_2[idx1] = (vec_1[idx1 & 0xFFFFFFDF] + (int64)omega_pow_use_rank[(idx1 & 0x3F) | 0x40] * vec_1[idx1 | 0x00000020]) % 8380417;
    vec_2[idx2] = (vec_1[idx2 & 0xFFFFFFDF] + (int64)omega_pow_use_rank[(idx2 & 0x3F) | 0x40] * vec_1[idx2 | 0x00000020]) % 8380417;
    vec_2[idx3] = (vec_1[idx3 & 0xFFFFFFDF] + (int64)omega_pow_use_rank[(idx3 & 0x3F) | 0x40] * vec_1[idx3 | 0x00000020]) % 8380417;
    vec_2[idx4] = (vec_1[idx4 & 0xFFFFFFDF] + (int64)omega_pow_use_rank[(idx4 & 0x3F) | 0x40] * vec_1[idx4 | 0x00000020]) % 8380417;
    vec_2[idx5] = (vec_1[idx5 & 0xFFFFFFDF] + (int64)omega_pow_use_rank[(idx5 & 0x3F) | 0x40] * vec_1[idx5 | 0x00000020]) % 8380417;
    vec_2[idx6] = (vec_1[idx6 & 0xFFFFFFDF] + (int64)omega_pow_use_rank[(idx6 & 0x3F) | 0x40] * vec_1[idx6 | 0x00000020]) % 8380417;
    vec_2[idx7] = (vec_1[idx7 & 0xFFFFFFDF] + (int64)omega_pow_use_rank[(idx7 & 0x3F) | 0x40] * vec_1[idx7 | 0x00000020]) % 8380417;

    vec_1[idx0] = (vec_2[idx0 & 0xFFFFFFBF] + (int64)omega_pow_use_rank[(idx0 & 0x7F) | 0x80] * vec_2[idx0 | 0x00000040]) % 8380417;
    vec_1[idx1] = (vec_2[idx1 & 0xFFFFFFBF] + (int64)omega_pow_use_rank[(idx1 & 0x7F) | 0x80] * vec_2[idx1 | 0x00000040]) % 8380417;
    vec_1[idx2] = (vec_2[idx2 & 0xFFFFFFBF] + (int64)omega_pow_use_rank[(idx2 & 0x7F) | 0x80] * vec_2[idx2 | 0x00000040]) % 8380417;
    vec_1[idx3] = (vec_2[idx3 & 0xFFFFFFBF] + (int64)omega_pow_use_rank[(idx3 & 0x7F) | 0x80] * vec_2[idx3 | 0x00000040]) % 8380417;
    vec_1[idx4] = (vec_2[idx4 & 0xFFFFFFBF] + (int64)omega_pow_use_rank[(idx4 & 0x7F) | 0x80] * vec_2[idx4 | 0x00000040]) % 8380417;
    vec_1[idx5] = (vec_2[idx5 & 0xFFFFFFBF] + (int64)omega_pow_use_rank[(idx5 & 0x7F) | 0x80] * vec_2[idx5 | 0x00000040]) % 8380417;
    vec_1[idx6] = (vec_2[idx6 & 0xFFFFFFBF] + (int64)omega_pow_use_rank[(idx6 & 0x7F) | 0x80] * vec_2[idx6 | 0x00000040]) % 8380417;
    vec_1[idx7] = (vec_2[idx7 & 0xFFFFFFBF] + (int64)omega_pow_use_rank[(idx7 & 0x7F) | 0x80] * vec_2[idx7 | 0x00000040]) % 8380417;

    vec_2[idx0] = (vec_1[idx0 & 0xFFFFFF7F] + (int64)omega_pow_use_rank[idx0 | 0x100] * vec_1[idx0 | 0x00000080]) % 8380417;
    vec_2[idx1] = (vec_1[idx1 & 0xFFFFFF7F] + (int64)omega_pow_use_rank[idx1 | 0x100] * vec_1[idx1 | 0x00000080]) % 8380417;
    vec_2[idx2] = (vec_1[idx2 & 0xFFFFFF7F] + (int64)omega_pow_use_rank[idx2 | 0x100] * vec_1[idx2 | 0x00000080]) % 8380417;
    vec_2[idx3] = (vec_1[idx3 & 0xFFFFFF7F] + (int64)omega_pow_use_rank[idx3 | 0x100] * vec_1[idx3 | 0x00000080]) % 8380417;
    vec_2[idx4] = (vec_1[idx4 & 0xFFFFFF7F] + (int64)omega_pow_use_rank[idx4 | 0x100] * vec_1[idx4 | 0x00000080]) % 8380417;
    vec_2[idx5] = (vec_1[idx5 & 0xFFFFFF7F] + (int64)omega_pow_use_rank[idx5 | 0x100] * vec_1[idx5 | 0x00000080]) % 8380417;
    vec_2[idx6] = (vec_1[idx6 & 0xFFFFFF7F] + (int64)omega_pow_use_rank[idx6 | 0x100] * vec_1[idx6 | 0x00000080]) % 8380417;
    vec_2[idx7] = (vec_1[idx7 & 0xFFFFFF7F] + (int64)omega_pow_use_rank[idx7 | 0x100] * vec_1[idx7 | 0x00000080]) % 8380417;
}